#include <common.cuh>
#include <new.h>
#include <chrono>
#include <sstream>

float* a_h;
float* b_h; 
float* out_h;
float* a_d;
float* b_d;
float* out_d;

auto main() -> int {
    a_h = (float*)malloc(N * sizeof(float));
    b_h = (float*)malloc(N * sizeof(float));
    out_h = (float*)malloc(N * sizeof(float));
    for (size_t i = 0; i < N; i++) {
        a_h[i] = b_h[i] = 1.0f;
        out_h[i] = 0;
    }

    hipMalloc(&a_d, N * sizeof(float));
    hipMalloc(&b_d, N * sizeof(float));
    hipMalloc(&out_d, N * sizeof(float));
    hipMemcpy(a_d, a_h, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out_h, N * sizeof(float), hipMemcpyHostToDevice);


    auto now = std::chrono::high_resolution_clock::now();
    // for (int i = 0; i < 100000; i++) {
    //     add_cpu(a_h, b_h, out_h, N);
    // }
    auto now_h = std::chrono::high_resolution_clock::now();
    std::stringstream ss;
    // ss << std::chrono::duration_cast<std::chrono::milliseconds>(now_h - now) << std::endl;

    now = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < 30000; i++) {
        sum<<<GRID_DIM, BLOCK_DIM>>>(a_d, out_d, N);
        hipDeviceSynchronize();
    }
    hipDeviceSynchronize();
    auto now_d = std::chrono::high_resolution_clock::now();
    ss << std::chrono::duration_cast<std::chrono::milliseconds>(now_d - now) << std::endl;
    ::printf("%s", ss.str().c_str());

    auto ret = 0.0f;
    hipMemset(out_d, 0, N * sizeof(float));
    sum<<<GRID_DIM, BLOCK_DIM>>>(a_d, out_d, N);
    hipDeviceSynchronize();
    hipMemcpy(out_h, out_d, N * sizeof(float), hipMemcpyDeviceToHost);
    for (size_t i = 0; i < BLOCK_DIM; i++) ret += out_h[i];
    ::printf("%f", ret);
}