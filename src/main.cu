#include <common.cuh>
#include <new.h>
#include <chrono>

constexpr size_t N = 1000000;
float* a_h;
float* b_h; 
float* out_h;
float* a_d;
float* b_d;
float* out_d;

auto main() -> int {
    a_h = (float*)malloc(N * sizeof(float));
    b_h = (float*)malloc(N * sizeof(float));
    out_h = (float*)malloc(N * sizeof(float));

    hipMalloc(&a_d, N * sizeof(float));
    hipMalloc(&b_d, N * sizeof(float));
    hipMalloc(&out_d, N * sizeof(float));

    print_hello<<<1, 1>>>();

    auto now = std::chrono::high_resolution_clock::now();
    auto now_h = std::chrono::high_resolution_clock::now();


    now = std::chrono::high_resolution_clock::now();
    auto now_d = std::chrono::high_resolution_clock::now();
}