#include "hip/hip_runtime.h"
#include <common.cuh>
#include <new.h>
#include <chrono>
#include <sstream>

auto main() -> int {
    float* a_h;
    float* out_h;
    float* a_d;
    float* out_d;

    a_h = (float*)malloc(N * sizeof(float));
    for (size_t i = 0; i < N; i++) {
        a_h[i] = 1.0f;
    }
    hipMalloc(&a_d, N * sizeof(float));
    hipMemcpy(a_d, a_h, N * sizeof(float), hipMemcpyHostToDevice);

    out_h = (float*)malloc((N / BLOCK_DIM) * sizeof(float)); 
    for (size_t i = 0; i < (N / BLOCK_DIM); i++) {
        out_h[i] = 0.0f;
    };
    hipMalloc(&out_d, (N / BLOCK_DIM) * sizeof(float));
    hipMemcpy(out_d, out_h, N * sizeof(float), hipMemcpyHostToDevice);

    auto now = std::chrono::high_resolution_clock::now();
    for (auto round = 0; round < 100; round++) {
        double ret_cpu = 0.0f;
        for (size_t i = 0; i < N; i++) {
            ret_cpu += a_h[i];
        }
        if (round == 99) [[unlikely]] ::printf("%f\n", ret_cpu);
    }
    auto now_after = std::chrono::high_resolution_clock::now();
    auto dur = std::chrono::duration_cast<std::chrono::milliseconds>(now_after - now).count();
    ::printf("%lldms\n", dur);

    now = std::chrono::high_resolution_clock::now();
    for (auto round = 0; round < 100; round++) {
        dim3 grid(N / BLOCK_DIM, 1);
        dim3 block(BLOCK_DIM, 1);
        reduce<<<grid, block>>>(a_d, out_d, N);
        hipMemcpy(out_h, out_d, (N / BLOCK_DIM) * sizeof(float), hipMemcpyDeviceToHost);
        double ret_cuda = 0.0f;
        for (size_t i = 0; i < (N / BLOCK_DIM); i++) ret_cuda += out_h[i];
        if (round == 99) [[unlikely]] ::printf("%f\n", ret_cuda);
    }
    now_after = std::chrono::high_resolution_clock::now();
    dur = std::chrono::duration_cast<std::chrono::milliseconds>(now_after - now).count();
    ::printf("%lldms\n", dur);
}