// main.cpp
// main.cpp

#include <algorithm>
#include <exception>
#include <type_traits>
#include <iostream>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/universal_vector.h>
#include <thrust/for_each.h>
#include <thrust/generate.h>
#include <thrust/iterator/counting_iterator.h>
#include <numeric>
#include <ranges>
#include <concepts>

#define DEVTID blockDim.x * blockIdx.x + threadIdx.x

auto main() -> int {
    // thrust::universal_vector<int> a;
    return 0;
}