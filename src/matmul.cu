#include "hip/hip_runtime.h"
#include <common.cuh>

// C = AB, the shape of A is (n, k), the shape of B is (k, m), so the shape of C is (n, m)
auto matmul_init_data(float** A_h, float** B_h, float** C_h, float** A_d, float** B_d, float** C_d) -> void {
    *A_h = (float*)malloc(N * K * sizeof(float));
    *B_h = (float*)malloc(K * M * sizeof(float));
    *C_h = (float*)malloc(N * M * sizeof(float));

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < K; j++) {
            (*A_h)[i * K + j] = (float)(i % 10);
        }
    }
    for (int i = 0; i < K; i++) {
        for (int j = 0; j < M; j++) {
            (*B_h)[i * M + j] = (float)(j % 10);
        }
    }
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            (*C_h)[i * M + j] = 0.0f;
        }
    }  
    
    hipMalloc(A_d, N * K * sizeof(float));
    hipMalloc(B_d, K * M * sizeof(float));
    hipMalloc(C_d, N * M * sizeof(float));
    hipMemcpy(*A_d, *A_h, N * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*B_d, *B_h, K * M * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*C_d, *C_h, N * M * sizeof(float), hipMemcpyHostToDevice);
}

auto matmul_cpu(float* a, float* b, float* c, size_t M, size_t K, size_t N) -> void {
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            float tmp = 0.0f;
            for (int s = 0; s < K; s++) {
                tmp += a[i * K + s] * b[s * N + j];
            }
            c[i * N + j] = tmp;
        }
    }
}

// __global__ auto matmul_kernel(float* a, float* b, float* c, size_t M, size_t K, size_t N) -> void {
//     int row = blockDim.x * blockIdx.x + threadIdx.x;
//     int col = blockDim.y * blockIdx.y + threadIdx.y;
// }