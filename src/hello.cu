#include "hip/hip_runtime.h"
#include <cstdio>
#include <common.cuh>

__global__ auto print_hello() -> void {
    ::printf("tid: %d, bid: %d: Hello!\n", threadIdx.x, blockIdx.x);
}

// auto main() -> int {
//     print_hello<<<1, 1>>>();
// }