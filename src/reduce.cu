#include "hip/hip_runtime.h"
#include <common.cuh>

__global__ auto reduce(float const* a, float* sum, size_t n) -> void {
    // for (auto i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += gridDim.x * blockDim.x) {
    //     atomicAdd(sum, a[i]);
    // }
    __shared__ float local[BLOCK_DIM];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    local[tid] = a[i];
    __syncthreads();

    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            local[tid] += local[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) sum[blockIdx.x] = local[tid];
}