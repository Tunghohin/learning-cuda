#include "hip/hip_runtime.h"
#include <common.cuh>

__global__ auto reduce_kernel(float const* a, float* sum, size_t n) -> void {
    __shared__ float local[BLOCK_DIM];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    local[tid] = a[i];
    __syncthreads();

    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            local[tid] += local[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) sum[blockIdx.x] = local[tid];
}