#include "hip/hip_runtime.h"
#include <common.cuh>

__host__ auto add_cpu(float const* a, float const* b, float* out, size_t n) -> void {
    for (int i = 0; i < n; i++) {
        out[i] = a[i] + b[i];
    }
}

__global__ auto add_kernel(float const* a, float const* b, float* out, size_t n) -> void {
    for (auto i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        out[i] = a[i] + b[i];
    }
}